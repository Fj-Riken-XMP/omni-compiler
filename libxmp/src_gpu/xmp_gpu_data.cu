#include "xmp_gpu_internal.h"

extern "C" void _XMP_gpu_init_gpudata_NOT_ALIGNED(_XMP_gpudata_t **host_desc, _XMP_gpudata_t **device_desc, void **device_addr, void *addr, size_t size) {
  _XMP_gpudata_t *host_d = NULL;
  _XMP_gpudata_t *device_d = NULL;

  // alloc desciptors
  host_d = (_XMP_gpudata_t *)_XMP_alloc(sizeof(_XMP_gpudata_t));
  _XMP_gpu_alloc((void **)&device_d, sizeof(_XMP_gpudata_t));

  // init host descriptor
  host_d->is_aligned_array = false;

  host_d->host_addr = addr;
  _XMP_gpu_alloc(&(host_d->device_addr), size);

  host_d->device_gpudata_desc = device_d;
  host_d->device_array_desc = NULL;

  host_d->size = size;

  *host_desc = host_d;
  *device_desc = device_d;
  *device_addr = host_d->device_addr;

  // init device descriptor
  hipMemcpy(device_d, host_d, sizeof(_XMP_gpudata_t), hipMemcpyHostToDevice);
}

extern "C" void _XMP_gpu_init_gpudata_ALIGNED(_XMP_gpudata_t **host_gpudata_desc, _XMP_gpudata_t **device_gpudata_desc, void **device_addr, void *addr, _XMP_array_t *array_desc) {
  _XMP_gpudata_t *host_d = NULL;
  _XMP_gpudata_t *device_d = NULL;
  _XMP_array_t *device_a = NULL;

  size_t array_size = (array_desc->total_elmts) * (array_desc->type_size);
  size_t array_desc_size = sizeof(_XMP_array_t) + sizeof(_XMP_array_info_t) * (array_desc->dim - 1);

  // alloc desciptors
  host_d = (_XMP_gpudata_t *)_XMP_alloc(sizeof(_XMP_gpudata_t));
  _XMP_gpu_alloc((void **)&device_d, sizeof(_XMP_gpudata_t));
  _XMP_gpu_alloc((void **)&(device_a), array_desc_size);

  // init host descriptor
  host_d->is_aligned_array = true;

  host_d->host_addr = addr;
  _XMP_gpu_alloc(&(host_d->device_addr), array_size);

  host_d->device_gpudata_desc = device_d;
  host_d->device_array_desc = device_a;

  host_d->size = array_size;

  *host_gpudata_desc = host_d;
  *device_gpudata_desc = device_d;
  *device_addr = host_d->device_addr;

  // init device descriptor
  hipMemcpy(device_d, host_d, sizeof(_XMP_gpudata_t), hipMemcpyHostToDevice);
  hipMemcpy(device_a, array_desc, array_desc_size, hipMemcpyHostToDevice);
}

extern "C" void _XMP_gpu_finalize_gpudata(_XMP_gpudata_t *desc) {
  _XMP_gpu_free(desc->device_addr);
  _XMP_gpu_free(desc->device_gpudata_desc);

  if (desc->is_aligned_array) {
    _XMP_gpu_free(desc->device_array_desc);
  }

  _XMP_free(desc);
}
