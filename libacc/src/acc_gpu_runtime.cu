#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "acc_internal.h"
#include "acc_gpu_internal.h"
#define BUF_LEN 256

int _ACC_gpu_device_count;
//int _ACC_gpu_max_thread;
//int _ACC_gpu_max_block_dim_x;
//int _ACC_gpu_max_block_dim_y;
//int _ACC_gpu_max_block_dim_z;

static int current_device_num = 0;
static void init_device(int dev_num);
static void finalize_device(int dev_num);

typedef struct acc_context{
  char isInitialized;
  void *stream_map;
  void *mpool;
}acc_context;

acc_context *contexts;


void _ACC_gpu_init(void) {
  hipError_t cuda_err;
  int i;

  cuda_err = hipGetDeviceCount(&_ACC_gpu_device_count);
  if(cuda_err == hipErrorNoDevice){
    _ACC_fatal("no GPU device");
  }else if(cuda_err == hipErrorInsufficientDriver){
    _ACC_fatal("installed CUDA driver is older than CUDA runtime library");
  }else if(cuda_err != hipSuccess){
    _ACC_gpu_fatal(cuda_err);
  }

  _ACC_DEBUG("Total number of GPUs = %d\n", _ACC_gpu_device_count)

  contexts = (acc_context*)_ACC_alloc(sizeof(acc_context) * _ACC_gpu_device_count);
  for(i = 0; i< _ACC_gpu_device_count; i++){
	contexts[i].isInitialized = 0;
	contexts[i].stream_map = NULL;
	contexts[i].mpool = NULL;
  }

  // //init each GPU
  // for(i=0;i<_ACC_gpu_device_count;i++){
  //   init_device(i);
  // }

  _ACC_gpu_set_device_num(0); //set device to default
}

void _ACC_gpu_finalize(void) {
  //finalize each GPU
  for(int i=0;i<_ACC_gpu_device_count;i++){
	if(contexts[i].isInitialized){
	  _ACC_gpu_set_device_num(i+1);
	  finalize_device(i);
	}
  }

  _ACC_free(contexts);
}

int _ACC_gpu_get_num_devices()
{
  int count;
  hipError_t error = hipGetDeviceCount(&count);
  if(error != hipSuccess){
    _ACC_gpu_fatal(error);
  }
  return count;
}

static void init_device(int dev_num){ //0-based
  hipError_t cuda_err;
  _ACC_DEBUG("initializing GPU %d\n",dev_num)

  if (hipSetDevice(dev_num) != hipSuccess) {
    _ACC_fatal("fail to set GPU device");
  }
  int *dummy;
  _ACC_gpu_alloc((void **)&dummy, sizeof(int));
  _ACC_gpu_free(dummy);
  
  // if(hipDeviceReset() != hipSuccess){
  //   _ACC_fatal("failed to reset GPU");
  // }

  hipDeviceProp_t dev_prop;
  cuda_err = hipGetDeviceProperties(&dev_prop, dev_num);
  if(cuda_err != hipSuccess){
    _ACC_fatal("fail to get GPU device properties");
  }
  _ACC_DEBUG("name : %s\n", dev_prop.name)
  _ACC_DEBUG("clock : %dKHz\n", dev_prop.clockRate)
  _ACC_DEBUG("cc : %d.%d\n",dev_prop.major, dev_prop.minor)
  _ACC_DEBUG("#sm : %d\n",dev_prop.multiProcessorCount)


  //init mpool
  contexts[dev_num].isInitialized = 1;
  contexts[dev_num].mpool = _ACC_gpu_mpool_init();
  //init stream hashmap
  contexts[dev_num].stream_map = _ACC_gpu_init_stream_map(16);
}

static void finalize_device(int dev_num){
  
  //finalize stream hashmap for previous device
  acc_context cont = contexts[dev_num];
  //printf("finalize_map(%d, %p)\n", dev_num, cont.stream_map);
  if(contexts[dev_num].isInitialized){
	_ACC_gpu_finalize_stream_map(cont.stream_map);
	_ACC_gpu_mpool_finalize(cont.mpool);
  }
  contexts[dev_num].isInitialized = 0;
}

void _ACC_gpu_set_device_num(int num)
{
  /* num is 1-origin */
  _ACC_DEBUG("device_num(%d)\n",num)
  hipError_t cuda_err;

  if(num < 0 || num > _ACC_gpu_device_count){
    _ACC_fatal("invalid device num in _ACC_gpu_set_device_num");
  }

  if(num == 0){ // 0 means default device num
    current_device_num = 0;
  }else{
    current_device_num = num - 1;
  }

  if (hipSetDevice(current_device_num) != hipSuccess) {
    _ACC_fatal("fail to set GPU device in _ACC_gpu_set_device_num");
  }
  
  // acc_context cont = contexts[current_device_num];
  // _ACC_gpu_set_stream_map(cont.stream_map);
  // _ACC_gpu_mpool_set(cont.mpool);

}

int _ACC_gpu_get_device_num(){
  return current_device_num + 1;
}

void _ACC_gpu_init_device_if_not_inited(int num) //0-based
{
  if(! contexts[num].isInitialized){
	init_device(num);
  }
}

void* _ACC_gpu_get_current_stream_map()
{
  _ACC_DEBUG("get_current_stream_map\n")
  void *stream_map = contexts[current_device_num].stream_map;
  if(stream_map == NULL){
	_ACC_gpu_init_device_if_not_inited(current_device_num);
  }
  return contexts[current_device_num].stream_map;
}

void* _ACC_gpu_get_current_mpool()
{
  _ACC_DEBUG("get_current_mpool\n")
  void *mpool = contexts[current_device_num].mpool;
  if(mpool == NULL){
	_ACC_gpu_init_device_if_not_inited(current_device_num);
  }
  return contexts[current_device_num].mpool;
}
