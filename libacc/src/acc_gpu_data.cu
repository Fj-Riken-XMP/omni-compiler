#include <stdio.h>
#include <stdarg.h>
#include "acc_internal.h"
#include "acc_gpu_internal.h"
#include "acc_gpu_data_struct.h"

static void register_memory(void *host_addr, size_t size);
static void unregister_memory(void *host_addr);

void _ACC_gpu_init_data(_ACC_gpu_data_t **host_data_desc, void **device_addr, void *addr, size_t offset, size_t size) {
  _ACC_gpu_data_t *host_data_d = NULL;

  // alloc desciptors
  host_data_d = (_ACC_gpu_data_t *)_ACC_alloc(sizeof(_ACC_gpu_data_t));

  // init host descriptor
  host_data_d->host_addr = addr;

  _ACC_gpu_alloc(&(host_data_d->device_addr), size);
  //host_data_d->host_array_desc = NULL;
  //host_data_d->device_array_desc = NULL;
  host_data_d->offset = offset;
  host_data_d->size = size;
  host_data_d->is_original = true;

  
  // about pagelock
  unsigned int flags;
  hipHostGetFlags(&flags, addr);
  hipError_t error = hipGetLastError();
  if(error == hipSuccess){
    //printf("memory is pagelocked\n");
    host_data_d->is_pagelocked = true;
  }else{
    //printf("memory is not pagelocked\n");
    host_data_d->is_pagelocked = false;
  }
  host_data_d->is_registered = false;
  
  
  // init params
  *host_data_desc = host_data_d;
  *device_addr = (void *)((char*)(host_data_d->device_addr) - offset);

  
  _ACC_gpu_add_data(host_data_d);
}

void _ACC_gpu2_init_data(_ACC_gpu_data_t **host_data_desc, void **device_addr, void *addr, size_t type_size, int dim, ...){

  va_list args;
  int i;
  _ACC_gpu_data_t *host_data_d = NULL;

  // alloc desciptors
  host_data_d = (_ACC_gpu_data_t *)_ACC_alloc(sizeof(_ACC_gpu_data_t));

  // init host descriptor
  host_data_d->host_addr = addr;
  host_data_d->type_size = type_size;
  host_data_d->dim = dim;

  // set array info
  _ACC_gpu_array_t *array_info = (_ACC_gpu_array_t *)_ACC_alloc(dim * sizeof(_ACC_gpu_array_t));
  va_start(args, dim);
  for(i=0;i<dim;i++){
    array_info[i].dim_offset = va_arg(args, int);
    if(i != 0 && array_info[i].dim_offset != 0){
      _ACC_fatal("Non-zero lower is allowed only top dimension");
    }
    array_info[i].dim_elmnts = va_arg(args, int);
  }
  va_end(args);
  int accumulation = 1;
  for(i=dim-1; i >= 0; i--){
    array_info[i].dim_acc = accumulation;
    accumulation *= array_info[i].dim_elmnts;
  }
  host_data_d->array_info = array_info;
  size_t size = accumulation * type_size;
  size_t offset = array_info[0].dim_offset * array_info[0].dim_acc * type_size;

  //device memory alloc
  _ACC_gpu_alloc(&(host_data_d->device_addr), size);
  host_data_d->offset = offset;
  host_data_d->size = size;
  host_data_d->is_original = true;

  //printf("hostaddr=%p, size=%zu, offset=%zu\n", addr, size, offset);
  // about pagelock
  unsigned int flags;
  hipHostGetFlags(&flags, addr);
  hipError_t error = hipGetLastError();
  if(error == hipSuccess){
    //printf("memory is pagelocked\n");
    host_data_d->is_pagelocked = true;
  }else{
    //printf("memory is not pagelocked\n");
    host_data_d->is_pagelocked = false;
  }
  host_data_d->is_registered = false;

  // init params
  *host_data_desc = host_data_d;
  *device_addr = (void *)((char*)(host_data_d->device_addr) - offset);

  _ACC_gpu_add_data(host_data_d);
}


void _ACC_gpu_pinit_data(_ACC_gpu_data_t **host_data_desc, void **device_addr, void *host_addr, size_t offset, size_t size) {
  _ACC_gpu_data_t *host_data_d = NULL;

  _ACC_gpu_data_t *present_host_data_desc;
  void *present_device_addr;
  unsigned char is_present = 0;
  _ACC_gpu_get_data_sub(&present_host_data_desc, &present_device_addr, host_addr, offset, size);
  if(present_host_data_desc != NULL){
    is_present = 1;
  }


  // alloc desciptor
  host_data_d = (_ACC_gpu_data_t *)_ACC_alloc(sizeof(_ACC_gpu_data_t));

  // init host descriptor
  host_data_d->host_addr = host_addr;
  host_data_d->offset = offset;
  host_data_d->size = size;
  if(is_present){
    host_data_d->device_addr = (void *)((char*)(present_device_addr) + offset); //is it correct? 
    host_data_d->is_original = false;
  }else{
    _ACC_gpu_alloc(&(host_data_d->device_addr), size);
    host_data_d->is_original = true;
    _ACC_gpu_add_data(host_data_d);
  }    


  // about pagelock
  if(is_present){
    host_data_d->is_pagelocked = present_host_data_desc->is_pagelocked;
    host_data_d->is_registered = present_host_data_desc->is_registered;
  }else{
    unsigned int flags;
    hipHostGetFlags(&flags, host_addr);
    hipError_t error = hipGetLastError();
    if(error == hipSuccess){
      host_data_d->is_pagelocked = true;
    }else{
      host_data_d->is_pagelocked = false;
    }
    host_data_d->is_registered = false;
  }

  
  
  // init params
  *host_data_desc = host_data_d;
  *device_addr = (void *)((char*)(host_data_d->device_addr) - offset);
}

void _ACC_gpu_finalize_data(_ACC_gpu_data_t *desc) {
  if(desc->is_original == true){
    if(desc->is_registered == true){
      unregister_memory(desc->host_addr);
    }

    if(_ACC_gpu_remove_data(desc) == false){
      _ACC_fatal("can't remove data from data table\n");
    }
    _ACC_gpu_free(desc->device_addr);
    //desc->device_addr = NULL;
  }

  _ACC_free(desc);
}

void _ACC_gpu_copy_data(_ACC_gpu_data_t *desc, size_t offset, size_t size, int direction){
  _ACC_gpu_copy((void*)((char*)(desc->host_addr) + offset), (void*)((char *)(desc->device_addr) + offset - desc->offset), size, direction);
}

void _ACC_gpu2_copy_data_using_pack(_ACC_gpu_data_t *desc, int direction, int isAsync, int *trans_info){
  int i;
  int dim = desc->dim;
  void *dev_buf;
  void *host_buf = NULL;
  int *info_length = trans_info + dim;
  const char useAsync = 0;

  int total_elmnts = 1;
  for(i=0;i<dim;i++){
    total_elmnts *= info_length[i];
  }

  size_t buf_size = total_elmnts * desc->type_size;
  //alloc buffer
  if(useAsync){
    if(host_buf == NULL){
      hipHostMalloc((void**)&host_buf, sizeof(double)*1024*1024);
    }
  }else{
    host_buf = (void *)_ACC_alloc( buf_size);
  }

  void *mpool;
  long long mpool_pos = 0;
  _ACC_gpu_mpool_get(&mpool);
  _ACC_gpu_mpool_alloc((void**)&dev_buf, buf_size, mpool, &mpool_pos);
  //alloc and copy of trans_info
  int *dev_trans_info;
  size_t trans_info_size = desc->dim * 3 * sizeof(int);
  _ACC_gpu_mpool_alloc((void**)&dev_trans_info, trans_info_size, mpool, &mpool_pos);
  _ACC_gpu_copy(trans_info, dev_trans_info, trans_info_size, 400);


  if(direction == 400){
    //host to device
    _ACC_gpu_pack_data_host(host_buf, desc->host_addr, desc->dim, total_elmnts, desc->type_size, trans_info);
    if(useAsync){
      hipMemcpyAsync(dev_buf, host_buf, buf_size, hipMemcpyHostToDevice);
    }else{
      _ACC_gpu_copy(host_buf, dev_buf, buf_size, 400);
    }
    
    void *dev_data = (void*)((char*)(desc->device_addr) - desc->offset);
    _ACC_gpu_unpack_data(dev_data, dev_buf, desc->dim, total_elmnts, desc->type_size, dev_trans_info);

    hipDeviceSynchronize();
  }else{
    //device to host
    void *dev_data = (void*)((char*)(desc->device_addr) - desc->offset);
    _ACC_gpu_pack_data(dev_buf, dev_data, desc->dim, total_elmnts, desc->type_size, dev_trans_info);
    if(useAsync){
      hipMemcpyAsync(host_buf, dev_buf, buf_size, hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }else{
      _ACC_gpu_copy(host_buf, dev_buf, buf_size, 401);
    }
    _ACC_gpu_unpack_data_host(desc->host_addr, host_buf, desc->dim, total_elmnts, desc->type_size, trans_info);
  }

  //free buffer
  _ACC_gpu_mpool_free(dev_buf, mpool);
  _ACC_gpu_mpool_free(dev_trans_info, mpool);

  if(! useAsync){
    _ACC_free(host_buf);
  }
}
  
void _ACC_gpu2_copy_subdata(_ACC_gpu_data_t *desc, int direction, int asyncId, ...){
  int dim = desc->dim;
  int *trans_info = (int *)_ACC_alloc(dim * 3 * sizeof(int));
  int *info_lower = trans_info;
  int *info_length = trans_info + dim;
  int *info_dim_acc = trans_info + dim*2;
  int i;

  va_list args;
  va_start(args, asyncId);
  for(i=0;i<dim;i++){
    info_lower[i] = va_arg(args, int);
    info_length[i] = va_arg(args, int);
    info_dim_acc[i] = desc->array_info[i].dim_acc;
  }
  va_end(args);

  char use_packing = 0;
  {
    for(i=0;i<dim;i++) if(info_length[i] != 1) break; //skip dims that len == 1
    if(i != dim){
      for(++i; i<dim; i++) if(info_lower[i] != 0 || info_length[i] != desc->array_info[i].dim_elmnts) break;
      if(i != dim){
	use_packing = 1;
      }
    }
  }

  if(use_packing){ //pack
    _ACC_gpu2_copy_data_using_pack(desc, direction, asyncId, trans_info);
  }else{
    int total_elmnts = 1;
    int offset_elmnts = 0;
    for(int i = 0; i<dim;i++){
      total_elmnts *= info_length[i];
      offset_elmnts += info_lower[i] * desc->array_info[i].dim_acc;
    }

    //printf("total_elmnts=%d, offset_el = %d\n", total_elmnts, offset_elmnts);
    size_t offset = offset_elmnts * desc->type_size;
    size_t size = total_elmnts * desc->type_size;
    _ACC_gpu_copy((void*)((char*)(desc->host_addr) + offset), (void*)((char *)(desc->device_addr) + offset - desc->offset), size, direction);
  }
}

void _ACC_gpu2_copy_data(_ACC_gpu_data_t *desc, int direction, int asyncId)
{
  switch(asyncId){
  case ACC_ASYNC_SYNC:
    _ACC_gpu_copy((void*)((char*)(desc->host_addr)), (void*)((char *)(desc->device_addr) - desc->offset), desc->size, direction);
    break;
  case ACC_ASYNC_NOVAL:
    _ACC_gpu_copy_async_all((void*)((char*)(desc->host_addr)), (void*)((char *)(desc->device_addr) - desc->offset), desc->size, direction);
    break;
  default:
    _ACC_gpu_copy_async((void*)((char*)(desc->host_addr)), (void*)((char *)(desc->device_addr) - desc->offset), desc->size, direction, asyncId);
  }
}

void _ACC_gpu_copy_data_async_all(_ACC_gpu_data_t *desc, int direction){
  //printf("_ACC_gpu_copy_data_async_all\n");

  //pagelock if data is not pagelocked
  if(desc->is_pagelocked == false && desc->is_registered == false){
    register_memory(desc->host_addr, desc->size);
    desc->is_registered = true;
  }

  _ACC_gpu_copy_async_all(desc->host_addr, desc->device_addr, desc->size, direction);
}


void _ACC_gpu_copy_data_async(_ACC_gpu_data_t *desc, int direction, int id){
  //printf("_ACC_gpu_copy_data_async\n");

  //pagelock if data is not pagelocked
  if(desc->is_pagelocked == false && desc->is_registered == false){
    register_memory(desc->host_addr, desc->size);
    desc->is_registered = true;
  }

  _ACC_gpu_copy_async(desc->host_addr, desc->device_addr, desc->size, direction, id);
}

void _ACC_gpu_copy_data_async_default(_ACC_gpu_data_t *desc, size_t offset, size_t size, int direction){
  //pagelock if data is not pagelocked
  if(desc->is_pagelocked == false && desc->is_registered == false){
    register_memory((void*)((char*)(desc->host_addr) + desc->offset), desc->size);
    desc->is_registered = true;
  }

  _ACC_gpu_copy_async_all((void*)((char*)(desc->host_addr) + offset), (void*)((char *)(desc->device_addr) + offset - desc->offset), size, direction);
}

void _ACC_gpu_find_data(_ACC_gpu_data_t **host_data_desc, void **device_addr, void *addr, size_t offset, size_t size) {
  //printf("finding data addr=%p, offset=%zu, size=%zu\n", addr, offset, size);
  _ACC_gpu_get_data_sub(host_data_desc, device_addr, addr, offset, size);
  if(*host_data_desc==NULL){
    _ACC_fatal("data not found");
  }
}

void _ACC_gpu2_find_data(_ACC_gpu_data_t **host_data_desc, void **device_addr, void *addr, size_t type_size, int dim, ...){
  int i;
  va_list args;

  // set array info
  _ACC_gpu_array_t *array_info = (_ACC_gpu_array_t *)_ACC_alloc(dim * sizeof(_ACC_gpu_array_t));
  va_start(args, dim);
  for(i=0;i<dim;i++){
    array_info[i].dim_offset = va_arg(args, int);
    if(i != 0 && array_info[i].dim_offset != 0){
      _ACC_fatal("Non-zero lower is allowed only top dimension");
    }
    array_info[i].dim_elmnts = va_arg(args, int);
  }
  va_end(args);
  
  int accumulation = 1;
  for(i=dim-1; i >= 0; i--){
    array_info[i].dim_acc = accumulation;
    accumulation *= array_info[i].dim_elmnts;
  }
  size_t size = accumulation * type_size;
  size_t offset = array_info[0].dim_offset * array_info[0].dim_acc * type_size;

  _ACC_gpu_get_data_sub(host_data_desc, device_addr, addr, offset, size);
  if(*host_data_desc==NULL){
    _ACC_fatal("gpu2 data not found");
  }
  (*host_data_desc)->array_info = array_info;
  (*host_data_desc)->type_size = type_size;
  (*host_data_desc)->dim = dim;
}

static void register_memory(void *host_addr, size_t size){
  printf("register_memory\n");
  hipError_t cuda_err = hipHostRegister(host_addr, size, hipHostRegisterPortable);
  if(cuda_err != hipSuccess){
    _ACC_gpu_fatal(cuda_err);
  }
}

static void unregister_memory(void *host_addr){
  printf("unregister_memory\n");
  hipError_t cuda_err = hipHostUnregister(host_addr);
  if(cuda_err != hipSuccess){
    _ACC_gpu_fatal(cuda_err);
  }
}

void _ACC_gpu_pcopy_data(_ACC_gpu_data_t *desc, size_t offset, size_t size, int direction){
  if(desc->is_original == true){
    _ACC_gpu_copy((void*)((char*)(desc->host_addr) + offset), (void*)((char *)(desc->device_addr) + offset - desc->offset), size, direction);
  }
}


